#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "cnpy.h"
// #include "more_reuse.cuh"
// #include "half_param_165.cuh"
// #include "general_more_reuse.cuh"
#include "less_CTA.cuh"

#define IC_stride 32
#define OC_stride 32

// #define H 112
// #define W 112
// #define IC 32
// #define OC 32
// #define C 32

#define H 56
#define W 56
#define IC 128
#define OC 128
#define C 128

// #define H 28
// #define W 28
// #define IC 256
// #define OC 256
// #define C 256

// #define H 14
// #define W 14
// #define IC 512
// #define OC 512
// #define C 512

using namespace std;

int main(int argc, char const *argv[])
{
	int N = 1, output_tile_H = 4, output_tile_W = 4;

	// Block and grid size
	int threadx_num = 32;
	dim3 block(threadx_num, 4, 1);

	// 1D grid
	// int block_x = (int)(H / output_tile_H) * (int)(W / output_tile_W), block_y = 1;
	// 2D grid
	int block_x = (int)(H / output_tile_H), block_y = (int)(W / output_tile_W);

	dim3 grid(block_x, block_y, 1);
	printf("block x: %d, block_y: %d\n", block_x, block_y);

	// Shared memory size
	size_t inter_size = output_tile_H * output_tile_W * OC_stride * sizeof(float);
	size_t filter_1_size = IC_stride * OC_stride * sizeof(float);
	size_t shared_size = inter_size + filter_1_size;

	// Sizes
	size_t input_shape = N * H * W * IC;
	size_t filter_d_shape = 3 * 3 * IC * 1;
	size_t filter_1_shape = 1 * 1 * IC * OC;
	size_t output_shape = N * H * W * OC;

	// Filenames
	string folder_name = "../npy/depth_conv_" + to_string(N) + "_" + to_string(H) + "_" + to_string(W) + "_" + to_string(IC) + "_" + to_string(OC) + "_" + to_string(3) + "/";
	string input_name = folder_name + "input.npy";
	string filter_d_name = folder_name + "filter_d.npy";
	string filter_1_name = folder_name + "filter_1.npy";
	string output_name = folder_name + "output.npy";

	// Definitions of GPU arrays
	float *input, *filter_d, *filter_1, *output;
	hipMalloc((void**)&input, input_shape * sizeof(float));
	hipMalloc((void**)&filter_d, filter_d_shape * sizeof(float));
	hipMalloc((void**)&filter_1, filter_1_shape * sizeof(float));
	hipMalloc((void**)&output, output_shape * sizeof(float));

	// Load data and copy to GPU arrays
	float *tmp;

    cnpy::NpyArray input_npy = cnpy::npy_load(input_name);
    tmp = input_npy.data<float>();
    hipMemcpy(input, tmp, input_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_d_npy = cnpy::npy_load(filter_d_name);
    tmp = filter_d_npy.data<float>();
    hipMemcpy(filter_d, tmp, filter_d_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_1_npy = cnpy::npy_load(filter_1_name);
    tmp = filter_1_npy.data<float>();
    hipMemcpy(filter_1, tmp, filter_1_shape * sizeof(float), hipMemcpyHostToDevice);

    // Execute kernel
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float ms = 0;
	int repeatition = 1000;

    for (int i = 0; i < repeatition; i++) {
    	hipMemset(output, 0, output_shape * sizeof(float));
    	float tmp_t = 0.0;
    	hipEventRecord(start);

    	// more_reuse.cuh and previous
	    // DepthConvFused_2_kernel0<<<grid, block>>>(input, filter_d, filter_1, output);

	    // // half_param_165.cuh
	    // DepthConvFused_2_kernel0<<<grid, block, shared_size>>>(
	    // 	input,
	    // 	filter_d, filter_1,
	    // 	output,
	    // 	H, W, C, C_stride,
	    // 	output_tile_H, output_tile_W
	    // );

	    // general_more_reuse.cuh
	    // DepthConvFused_2_kernel0<<<grid, block, shared_size>>>(
	    // 	input, 
	    // 	filter_d, filter_1,
	    // 	output,
	    // 	H, W, C, C_stride
	    // );

	    // less_CTA.cuh
	    DepthConvFused_2_kernel0 <H, W, IC, OC, 
	    							IC_stride, OC_stride> <<<grid, block, shared_size>>> (
	    	input,
	    	filter_d, filter_1,
	    	output
	    );


	    hipEventRecord(stop);

	    hipEventSynchronize(stop);
		hipEventElapsedTime(&tmp_t, start, stop);
		ms += tmp_t / repeatition;
    }
    
	printf("Fusion running time is %f us.\n", ms * 1000);

    // Verification
    // Something wrong with float, use double and convert back
    cnpy::NpyArray output_npy = cnpy::npy_load(output_name);
    double *tmp2 = output_npy.data<double>();

    // Copy result back to CPU for comparison
    float *result;
    result = (float*)malloc(output_shape * sizeof(float));
    hipMemcpy(result, output, output_shape * sizeof(float), hipMemcpyDeviceToHost);
    int count = 0;
    for(int i = 0; i < output_shape; i++) {
    	// printf("%d, %f, %lf\n", i, result[i], tmp2[i]);
    	// assert(abs(result[i] - (float)tmp2[i]) < 1e-4);
    	if (abs(result[i] - (float)tmp2[i]) > 1e-3) {
    		printf("%d, %f, %lf\n", i, result[i], tmp2[i]);
    		count++;
    	}
    }
    printf("Wrong count: %d\n", count);

    hipFree(input);
    hipFree(filter_d);
    hipFree(filter_1);
    hipFree(output);

	return 0;
}