#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "cnpy.h"
#include "general_more_reuse.cuh"

using namespace std;

#define BLOCK_Y_SIZE 4 // Fix this value in a foreseeable future

int main(int argc, char const *argv[])
{
	// int N = 1, output_tile_H = 2, output_tile_W = 2;
	int N = 1, output_tile_H = 4, output_tile_W = 4;

	// int H = 112, W = 112, C = 32;
	int H = 56, W = 56, C = 128;
	// int H = 28, W = 28, C = 256;
	// int H = 14, W = 14, C = 512;

	// Block and grid size
	int threadx_num = 32, C_stride = 32;
	dim3 block(threadx_num, BLOCK_Y_SIZE, 1);

	// 1D grid
	// int block_x = (int)(H / output_tile_H) * (int)(W / output_tile_W), block_y = 1;
	// 2D grid
	int block_x = (int)(H / output_tile_H), block_y = (int)(W / output_tile_W);

	dim3 grid(block_x, block_y, 1);
	printf("block x: %d, block_y: %d\n", block_x, block_y);

	// Shared memory size
	size_t inter_size = output_tile_H * output_tile_W * C_stride * sizeof(float);
	size_t filter_1_size = C_stride * C_stride * sizeof(float);
	size_t shared_size = inter_size + filter_1_size;

	// Sizes
	size_t input_shape = N * H * W * C;
	size_t filter_d_shape = 3 * 3 * C * 1;
	size_t filter_1_shape = 1 * 1 * C * C;
	size_t output_shape = N * H * W * C;

	// Filenames
	string input_name = "../npy/depth_conv_input_" + to_string(N) + "_" + to_string(H) + "_" + to_string(W) + "_" + to_string(C) + ".npy";
	string filter_d_name = "../npy/depth_conv_filter_d_3_3_" + to_string(C) + "_1.npy";
	string filter_1_name = "../npy/depth_conv_filter_1_1_1_" + to_string(C) + "_" + to_string(C) + ".npy";
	string output_name = "../npy/depth_conv_output_" + to_string(N) + "_" + to_string(H) + "_" + to_string(W) + "_" + to_string(C) + ".npy";

	// Definitions of GPU arrays
	float *input, *filter_d, *filter_1, *output;
	hipMalloc((void**)&input, input_shape * sizeof(float));
	hipMalloc((void**)&filter_d, filter_d_shape * sizeof(float));
	hipMalloc((void**)&filter_1, filter_1_shape * sizeof(float));
	hipMalloc((void**)&output, output_shape * sizeof(float));

	// Load data and copy to GPU arrays
	float *tmp;

    cnpy::NpyArray input_npy = cnpy::npy_load(input_name);
    tmp = input_npy.data<float>();
    hipMemcpy(input, tmp, input_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_d_npy = cnpy::npy_load(filter_d_name);
    tmp = filter_d_npy.data<float>();
    hipMemcpy(filter_d, tmp, filter_d_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_1_npy = cnpy::npy_load(filter_1_name);
    tmp = filter_1_npy.data<float>();
    hipMemcpy(filter_1, tmp, filter_1_shape * sizeof(float), hipMemcpyHostToDevice);

    // Execute kernel
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float ms = 0;
	int repeatition = 1000;

    for (int i = 0; i < repeatition; i++) {
    	hipMemset(output, 0, output_shape * sizeof(float));
    	float tmp_t = 0.0;
    	hipEventRecord(start);

	    // DepthConvFused_2_kernel0<<<grid, block>>>(input, filter_d, filter_1, output);
	    DepthConvFused_2_kernel0<<<grid, block, shared_size>>>(
	    	input, 
	    	filter_d, filter_1, 
	    	output, 
	    	H, W, C, C_stride,
	    	output_tile_H, output_tile_W
	    );

	    hipEventRecord(stop);

	    hipEventSynchronize(stop);
		hipEventElapsedTime(&tmp_t, start, stop);
		ms += tmp_t / repeatition;
    }
    
	printf("Fusion running time is %f us.\n", ms * 1000);

    // Verification
    // Something wrong with float, use double and convert back
    cnpy::NpyArray output_npy = cnpy::npy_load(output_name);
    double *tmp2 = output_npy.data<double>();

    // Copy result back to CPU for comparison
    float *result;
    result = (float*)malloc(output_shape * sizeof(float));
    hipMemcpy(result, output, output_shape * sizeof(float), hipMemcpyDeviceToHost);
    int count = 0;
    for(int i = 0; i < output_shape; i++) {
    	// printf("%d, %f, %lf\n", i, result[i], tmp2[i]);
    	// assert(abs(result[i] - (float)tmp2[i]) < 1e-4);
    	if (abs(result[i] - (float)tmp2[i]) > 1e-3) {
    		printf("%d, %f, %lf\n", i, result[i], tmp2[i]);
    		count++;
    	}
    }
    printf("Wrong count: %d\n", count);

    hipFree(input);
    hipFree(filter_d);
    hipFree(filter_1);
    hipFree(output);

	return 0;
}