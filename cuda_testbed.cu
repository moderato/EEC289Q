#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "cnpy.h"
#include "kernel.cuh"

using namespace std;

int main(int argc, char const *argv[])
{
	// NHWC
	int N = 1;
	int H = 112, W = 112, C = 32;
	// int H = 56, W = 56, C = 128;
	// int H = 28, W = 28, C = 256;
	// int H = 14, W = 14, C = 512;

	// Sizes
	size_t input_shape = N * H * W * C;
	size_t filter_d_shape = 3 * 3 * C * 1;
	size_t filter_1_shape = 1 * 1 * C * C;
	size_t output_shape = N * H * W * C;

	// Filenames
	string input_name = "input_" + to_string(N) + "_" + to_string(H) + "_" + to_string(W) + "_" + to_string(C) + ".npy";
	string filter_d_name = "filter_d_3_3_" + to_string(C) + "_1.npy";
	string filter_1_name = "filter_1_1_1_" + to_string(C) + "_" + to_string(C) + ".npy";
	string output_name = "output_" + to_string(N) + "_" + to_string(H) + "_" + to_string(W) + "_" + to_string(C) + ".npy";

	// Definitions of GPU arrays
	float *input, *filter_d, *filter_1, *output;
	hipMalloc((void**)&input, input_shape * sizeof(float));
	hipMalloc((void**)&filter_d, filter_d_shape * sizeof(float));
	hipMalloc((void**)&filter_1, filter_1_shape * sizeof(float));
	hipMalloc((void**)&output, output_shape * sizeof(float));

	// Load data and copy to GPU arrays
	float *tmp;

    cnpy::NpyArray input_npy = cnpy::npy_load(input_name);
    tmp = input_npy.data<float>();
    hipMemcpy(input, tmp, input_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_d_npy = cnpy::npy_load(filter_d_name);
    tmp = filter_d_npy.data<float>();
    hipMemcpy(filter_d, tmp, filter_d_shape * sizeof(float), hipMemcpyHostToDevice);

    cnpy::NpyArray filter_1_npy = cnpy::npy_load(filter_1_name);
    tmp = filter_1_npy.data<float>();
    hipMemcpy(filter_1, tmp, filter_1_shape * sizeof(float), hipMemcpyHostToDevice);

    // Execute kernel
    dim3 block(32, 4, 1);
    dim3 grid(1, 3136, 1);
    GeneralFused_2_kernel0<<<grid, block>>>(input, filter_d, filter_1, output);
    hipDeviceSynchronize();

    // Verification
    // Something wrong with float, use double and convert back
    cnpy::NpyArray output_npy = cnpy::npy_load(output_name);
    double *tmp2 = output_npy.data<double>();

    // 
    float *result;
    result = (float*)malloc(output_shape * sizeof(float));
    hipMemcpy(result, output, output_shape * sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < output_shape; i++) {
    	assert(abs(result[i] - (float)tmp2[i]) < 1e-4);
    }

    hipFree(input);
    hipFree(filter_d);
    hipFree(filter_1);
    hipFree(output);

	return 0;
}